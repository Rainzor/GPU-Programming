#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include ""

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

//! Block size used for CUDA kernel launch. 
#define blockSize 32

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

//! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
//! These are called ping-pong buffers.
glm::vec3* dev_pos = nullptr;
glm::vec3* dev_vel1 = nullptr;
glm::vec3* dev_vel2 = nullptr;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int* dev_particleArrayIndices = nullptr; // What index in dev_pos and dev_velX represents this particle?
int* dev_particleGridIndices = nullptr; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;


int *dev_gridCellStartIndices = nullptr; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices = nullptr;   // to this cell?
int2* dev_gridCellRanges = nullptr;

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3* dev_pos_gathered = nullptr;
glm::vec3* dev_vel_gathered = nullptr;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellRanges, gridCellCount * sizeof(int) * 2);
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  
  hipMalloc((void**)&dev_pos_gathered, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos_gathered failed!");

  hipMalloc((void**)&dev_vel_gathered, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel_gathered failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids

  glm::vec3 center(0.0f, 0.0f, 0.0f);
  glm::vec3 separate(0.0f, 0.0f, 0.0f);
  glm::vec3 cohesion(0.0f, 0.0f, 0.0f);

  int rule1NumNeighbors = 0;
  int rule2NumNeighbors = 0;
  glm::vec3 res_vel = glm::vec3(0.0f, 0.0f, 0.0f);

  glm::vec3 thisPos = pos[iSelf];

  // compute the velocity change based on the three rules
  for (int i = 0; i < N; i++) {
    if (i == iSelf) {
      continue;
    }

    glm::vec3 otherPos = pos[i];
    float distance = glm::length(otherPos - thisPos);

    if (distance < rule1Distance) {
      center += otherPos;
      rule1NumNeighbors++;
    }

    if (distance < rule2Distance) {
      separate -= otherPos - thisPos;
    }

    if (distance < rule3Distance) {
      cohesion += vel[i];
      rule2NumNeighbors++;
    }
  }

  // update velocity
  if (rule1NumNeighbors > 0) {
    center /= rule1NumNeighbors;
    res_vel += (center - thisPos) * rule1Scale;
  }
  res_vel += separate * rule2Scale;
  if (rule2NumNeighbors > 0) {
    cohesion /= rule2NumNeighbors;
    res_vel += cohesion * rule3Scale;
  }


  return res_vel;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  //! Ping-pong the velocity buffers: avoid read and write conflicts, reduce latency
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. 
  //? Question: why NOT vel1?

  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  // Compute a new velocity based on pos and vel1
  glm::vec3 delta_vel = computeVelocityChange(N, index, pos, vel1);
  
  // Ping-pong the velocity buffers
  // Update the velocity2
  glm::vec3 new_vel = vel1[index] + delta_vel;
  // Clamp the speed
  float speed = glm::length(new_vel);
  if (speed > maxSpeed) {
    new_vel = new_vel * maxSpeed / speed;
  }

  vel2[index] = new_vel;


}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(const int N, const int gridResolution,
  const glm::vec3 gridMin, const float inverseCellWidth,
  const glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    int thisParticleIndex = threadIdx.x + (blockIdx.x * blockDim.x);
    if (thisParticleIndex >= N) {
      return;
    }
  
    glm::vec3 thisPos = pos[thisParticleIndex];
    glm::vec3 thisCellPos = (thisPos - gridMin) * inverseCellWidth;
    glm::ivec3 thisCellIndex3{ thisCellPos.x, thisCellPos.y, thisCellPos.z };
    int thisCellIndex = gridIndex3Dto1D(thisCellIndex3.x, thisCellIndex3.y, thisCellIndex3.z, gridResolution);

    indices[thisParticleIndex] = thisParticleIndex;
    gridIndices[thisParticleIndex] = thisCellIndex;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetInt2Buffer(int N, int2 *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = make_int2(value, value);
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int2 *gridCellRanges) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

  int particleIndex = threadIdx.x + (blockIdx.x * blockDim.x);
  if (particleIndex >= N) {
    return;
  }
  int gridIndex = particleGridIndices[particleIndex];
  if(particleIndex == 0){
    gridCellRanges[gridIndex].x = 0;
  }else{
    int prevGridIndex = particleGridIndices[particleIndex-1];
    if(gridIndex != prevGridIndex){
      gridCellRanges[gridIndex].x = particleIndex;
      gridCellRanges[prevGridIndex].y = particleIndex;
    }
    if(particleIndex == N-1){
      gridCellRanges[gridIndex].y = N;
    }
  }

}

__global__ void kernUpdateVelNeighborSearchScattered(
  const int N, const int gridResolution, const glm::vec3 gridMin,
  const float inverseCellWidth, const float cellWidth,
  const int2 *gridCellRanges,
  const int *particleArrayIndices,
  const glm::vec3 *pos,const glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  int thisParticleIndex = threadIdx.x + (blockIdx.x * blockDim.x);
  if (thisParticleIndex >= N) {
    return;
  }

  glm::vec3 thisPos = pos[thisParticleIndex];
  glm::vec3 thisCellPos = (thisPos - gridMin) * inverseCellWidth;
  glm::ivec3 thisCellIndex3{ thisCellPos.x, thisCellPos.y, thisCellPos.z };
  
  glm::vec3 absolutePos = thisCellPos - glm::floor(thisCellPos) - 0.5f;
  glm::ivec3 quadrant;
  quadrant.x = (absolutePos.x > 0) ? 1 : -1;
  quadrant.y = (absolutePos.y > 0) ? 1 : -1;
  quadrant.z = (absolutePos.z > 0) ? 1 : -1;

  int rule1NumNeighbors = 0;
  int rule2NumNeighbors = 0;
  
  glm::vec3 delta_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 center(0.0f, 0.0f, 0.0f);
  glm::vec3 separate(0.0f, 0.0f, 0.0f);
  glm::vec3 cohesion(0.0f, 0.0f, 0.0f);

  for(int k=0; k < 8; k++){
    glm::ivec3 offset {k & 1, (k & 2) >> 1, (k & 4) >> 2};
    glm::ivec3 neighborCellIndex3 = thisCellIndex3 + offset * quadrant;
    int neighborCellIndex = gridIndex3Dto1D(neighborCellIndex3.x, neighborCellIndex3.y, neighborCellIndex3.z, gridResolution);
    if (neighborCellIndex < 0 || neighborCellIndex >= gridResolution * gridResolution * gridResolution) {
      continue;
    }
    int2 range = gridCellRanges[neighborCellIndex];
    for (int i = range.x; i < range.y; i++) {
      int otherParticleIndex = particleArrayIndices[i];
      if (otherParticleIndex == thisParticleIndex) {
        continue;
      }
      glm::vec3 otherPos = pos[otherParticleIndex];
      float distance = glm::length(otherPos - thisPos);

      if(distance > cellWidth)
        continue;
      if (distance < rule1Distance) {
        center += otherPos;
        rule1NumNeighbors++;
      }
      if (distance < rule2Distance) {
        separate -= otherPos - thisPos;
      }
      if (distance < rule3Distance) {
        cohesion += vel1[otherParticleIndex];
        rule2NumNeighbors++;
      }
    }
  }

  if (rule1NumNeighbors > 0) {
    center /= rule1NumNeighbors;
    delta_vel += (center - thisPos) * rule1Scale;
  }

  delta_vel += separate * rule2Scale;

  if (rule2NumNeighbors > 0) {
    cohesion /= rule2NumNeighbors;
    delta_vel += cohesion * rule3Scale;
  }

  glm::vec3 new_vel = vel1[thisParticleIndex] + delta_vel;
  // Clamp the speed
  float speed = glm::length(new_vel);
  if (speed > maxSpeed) {
    new_vel = new_vel * maxSpeed / speed;
  }
  vel2[thisParticleIndex] = new_vel;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  const int N, const int gridResolution, const glm::vec3 gridMin,
  const float inverseCellWidth, const float cellWidth,
  const int2 *gridCellRanges,
  const glm::vec3 *pos,const glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  int thisParticleIndex = threadIdx.x + (blockIdx.x * blockDim.x);
  if (thisParticleIndex >= N) {
    return;
  }

  glm::vec3 thisPos = pos[thisParticleIndex];
  glm::vec3 thisCellPos = (thisPos - gridMin) * inverseCellWidth;
  glm::ivec3 thisCellIndex3{ thisCellPos.x, thisCellPos.y, thisCellPos.z };
  
  glm::vec3 absolutePos = thisCellPos - glm::floor(thisCellPos) - 0.5f;
  glm::ivec3 quadrant;
  quadrant.x = (absolutePos.x > 0) ? 1 : -1;
  quadrant.y = (absolutePos.y > 0) ? 1 : -1;
  quadrant.z = (absolutePos.z > 0) ? 1 : -1;

  int rule1NumNeighbors = 0;
  int rule2NumNeighbors = 0;
  
  glm::vec3 delta_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 center(0.0f, 0.0f, 0.0f);
  glm::vec3 separate(0.0f, 0.0f, 0.0f);
  glm::vec3 cohesion(0.0f, 0.0f, 0.0f);

  for(int k=0; k < 8; k++){
    glm::ivec3 offset {k & 1, (k & 2) >> 1, (k & 4) >> 2};
    glm::ivec3 neighborCellIndex3 = thisCellIndex3 + offset * quadrant;
    int neighborCellIndex = gridIndex3Dto1D(neighborCellIndex3.x, neighborCellIndex3.y, neighborCellIndex3.z, gridResolution);
    if (neighborCellIndex < 0 || neighborCellIndex >= gridResolution * gridResolution * gridResolution) {
      continue;
    }
    int2 range = gridCellRanges[neighborCellIndex];
    for (int i = range.x; i < range.y; i++) {
      glm::vec3 otherPos = pos[i];
      glm::vec3 otherVel = vel1[i];
      float distance = glm::length(otherPos - thisPos);

      if(distance > cellWidth || distance == 0)
        continue;
      if (distance < rule1Distance) {
        center += otherPos;
        rule1NumNeighbors++;
      }
      if (distance < rule2Distance) {
        separate -= otherPos - thisPos;
      }
      if (distance < rule3Distance) {
        cohesion += otherVel;
        rule2NumNeighbors++;
      }
    }
  }

  if (rule1NumNeighbors > 0) {
    center /= rule1NumNeighbors;
    delta_vel += (center - thisPos) * rule1Scale;
  }

  delta_vel += separate * rule2Scale;

  if (rule2NumNeighbors > 0) {
    cohesion /= rule2NumNeighbors;
    delta_vel += cohesion * rule3Scale;
  }

  glm::vec3 new_vel = vel1[thisParticleIndex] + delta_vel;
  // Clamp the speed
  float speed = glm::length(new_vel);
  if (speed > maxSpeed) {
    new_vel = new_vel * maxSpeed / speed;
  }
  vel2[thisParticleIndex] = new_vel;



}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  // Update the velocity first
  kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, dev_vel1, dev_vel2);


  // TODO-1.2 ping-pong the velocity buffers
  // Swap the velocity buffers
  std::swap(dev_vel1, dev_vel2);
  kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos, dev_vel1);

}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed

  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  // Label each particle with its array index as well as its grid index.
  kernComputeIndices << <fullBlocksPerGrid, blockSize >> >(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, 
      dev_pos, 
      dev_particleArrayIndices, dev_particleGridIndices);
  
  // Unstable key sort using Thrust
  dev_thrust_particleGridIndices = thrust::device_pointer_cast(dev_particleGridIndices);
  dev_thrust_particleArrayIndices = thrust::device_pointer_cast(dev_particleArrayIndices);
  thrust::sort_by_key(dev_thrust_particleGridIndices, 
                      dev_thrust_particleGridIndices + numObjects, 
                      dev_thrust_particleArrayIndices);

  // Reset the grid cell start and end indices
  kernResetInt2Buffer << <fullBlocksPerGrid, blockSize >> >(gridCellCount, dev_gridCellRanges, -1);

  // Identify the start point of each cell in the gridIndices array.
  kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_particleGridIndices, dev_gridCellRanges);

  // Update a boid's velocity using the uniform grid to reduce the number of boids that need to be checked.
  kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> >(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, 
      dev_gridCellRanges, dev_particleArrayIndices, 
      dev_pos, dev_vel1, dev_vel2);

  // Swap the velocity buffers as Ping-pong buffers
  std::swap(dev_vel1, dev_vel2);

  // Update the position
  kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos, dev_vel1);

}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  // Label each particle with its array index as well as its grid index.
  kernComputeIndices << <fullBlocksPerGrid, blockSize >> >(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, 
      dev_pos, 
      dev_particleArrayIndices, dev_particleGridIndices);
  
  // Unstable key sort using Thrust
  dev_thrust_particleGridIndices = thrust::device_pointer_cast(dev_particleGridIndices);
  dev_thrust_particleArrayIndices = thrust::device_pointer_cast(dev_particleArrayIndices);
  thrust::sort_by_key(dev_thrust_particleGridIndices, 
                      dev_thrust_particleGridIndices + numObjects, 
                      dev_thrust_particleArrayIndices);

  // Reset the grid cell start and end indices
  kernResetInt2Buffer << <fullBlocksPerGrid, blockSize >> >(gridCellCount, dev_gridCellRanges, -1);

  // Identify the start point of each cell in the gridIndices array.
  kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_particleGridIndices, dev_gridCellRanges);

  // gather the position and velocity data according to the sorted indices
  auto thrust_pos = thrust::device_pointer_cast(dev_pos);
  auto thrust_vel1 = thrust::device_pointer_cast(dev_vel1);
  auto thrust_pos_gathered = thrust::device_pointer_cast(dev_pos_gathered);
  auto thrust_vel_gathered = thrust::device_pointer_cast(dev_vel_gathered);

  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle
  // all the particle data in the simulation array.
  thrust::gather(dev_thrust_particleArrayIndices, dev_thrust_particleArrayIndices + numObjects, thrust_pos,
                 thrust_pos_gathered);
  thrust::gather(dev_thrust_particleArrayIndices, dev_thrust_particleArrayIndices + numObjects, thrust_vel1,
                 thrust_vel_gathered);

  // Update a boid's velocity using the uniform grid to reduce the number of boids that need to be checked.
  kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> >(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, 
      dev_gridCellRanges, 
      dev_pos_gathered, dev_vel_gathered, dev_vel2);

  // Swap the velocity buffers as Ping-pong buffers
  std::swap(dev_vel1, dev_vel2);
  std::swap(dev_pos, dev_pos_gathered);
  // Update the position
  kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos, dev_vel1);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellRanges);

  hipFree(dev_pos_gathered);
  hipFree(dev_vel_gathered);

  checkCUDAErrorWithLine("hipFree failed!");
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /* Exclusive scan in Block Level
        * the size of odata should be BLOCK_SIZE * gird_size, 
        * so that need to padding the last block
        * @parms
        * n: the number of elements in idata
        */
        __global__ void kernInclusiveScanPerBlock(int n, int* odata, const int* idata){
            int tid = threadIdx.x;
            int bid = blockIdx.x;
            int gid = bid * blockDim.x + tid;
            __shared__ int buffer[2][BLOCK_SIZE];
            int ping = 0, pong = 1;
            buffer[ping][tid] = (gid > 0 && gid < n) ? idata[gid - 1] : 0;
            __syncthreads();
            for(int d = 1; d < blockDim.x; d <<= 1){
                ping = 1 - ping;
                pong = 1 - pong;
                if(tid >= d){
                    buffer[ping][tid] = buffer[pong][tid] + buffer[pong][tid - d];
                }
                else{
                    buffer[ping][tid] = buffer[pong][tid];
                }
                __syncthreads();
            }
            odata[gid] = buffer[ping][tid];
        }
        

        /*
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // Malloc different level of memory
            int level = (ilog2ceil(n)+7)/8;
            int** dev_ptr = new int*[level];
            int* grid_size = new int[level];
            int len = n;
            for(int i = 0; i < level; i++){
                grid_size[i] = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
                hipMalloc((void**)&dev_ptr[i], grid_size[i] * BLOCK_SIZE * sizeof(int));
                checkCUDAError("hipMalloc dev_ptr failed!");
                hipMemset(dev_ptr[i], 0, grid_size[i] * BLOCK_SIZE * sizeof(int));
                len = grid_size[i];
            }
            int* dev_tempbuff;
            int temp_size = n;
            hipMalloc((void**)&dev_tempbuff, temp_size*sizeof(int));
            checkCUDAError("hipMalloc dev_tempbuff failed!");
            hipMemcpy(dev_tempbuff, idata, temp_size*sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata failed!");

            timer().startGpuTimer();

            // Scan each block in different level
            for(int i = 0; i < level; i++){
                kernInclusiveScanPerBlock<<<grid_size[i], BLOCK_SIZE>>>(temp_size, dev_ptr[i], dev_tempbuff);
                // Gather the last element of each block
                if(i < level - 1){
                    Common::kernExtractLastElementPerBlock<<<grid_size[i+1], BLOCK_SIZE>>>( grid_size[i], 
                                                                                    BLOCK_SIZE, 
                                                                                    dev_tempbuff, 
                                                                                    dev_ptr[i]);
                }
                temp_size = grid_size[i];
            }
            // Scatter the offset to the original array
            for(int i = level - 2; i >= 0; i--){
                Common::kernAddOffset<<<grid_size[i], BLOCK_SIZE>>>(grid_size[i] * BLOCK_SIZE, dev_ptr[i], dev_ptr[i+1]);
            }

            timer().endGpuTimer();

            // Copy the result to the host
            hipMemcpy(odata, dev_ptr[0], n*sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed!");
            
            // Free the memory
            for(int i = 0; i < level; i++){
                hipFree(dev_ptr[i]);
            }
            hipFree(dev_tempbuff);
            delete[] dev_ptr;
            delete[] grid_size;

        }
    }
}

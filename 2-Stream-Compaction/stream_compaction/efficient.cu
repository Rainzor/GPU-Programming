#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        /* Exclusive scan in Block Level using work-efficient algorithm O(n)
        * the size of odata should be BLOCK_SIZE * gird_size, 
        * so that need to padding the last block
        * @parms
        * n: the number of elements in idata
        */
        __global__ void kernInclusiveScanPerBlock(int n, int* odata, const int* idata){
            int tid = threadIdx.x;
            int tid2 = tid << 1;
            int gid2 = blockIdx.x * BLOCK_SIZE + tid2; // Each thread handles 2 elements
            int global_base = blockIdx.x * BLOCK_SIZE;
            __shared__ int buffer[BLOCK_SIZE+BLOCK_SIZE/NUM_BANKS];

            // Load the data to the shared memory
            int ai = tid;
            int bi = tid + BLOCK_SIZE/2;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            int gid = global_base + tid;
            buffer[ai] = (0 < gid && gid < n) ? idata[gid-1] : 0;
            gid = global_base + tid + BLOCK_SIZE/2;
            buffer[bi] = (0 < gid && gid < n) ? idata[gid-1] : 0;   
            __syncthreads();
            
            // Up-Sweep (Reduce)
            int offset = 1;
            for(int stride = blockDim.x; stride > 0; stride >>= 1){
                if(tid < stride){
                    int ai = offset * (tid2 + 1) - 1;
                    int bi = offset * (tid2 + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    buffer[bi] += buffer[ai];
                }
                offset <<= 1;
                __syncthreads();
            }

            // Down-Sweep (Distribute)
            int sum = 0;
            if(tid == blockDim.x - 1){
                ai = BLOCK_SIZE - 1 + CONFLICT_FREE_OFFSET(BLOCK_SIZE - 1);
                sum = buffer[ai];
                buffer[ai] = 0;
            }
            __syncthreads();

            offset = blockDim.x;
            for(int stride = 1; stride <= blockDim.x; stride <<= 1){
                if(tid < stride){
                    int ai = offset * (tid2 + 1) - 1;
                    int bi = offset * (tid2 + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    int t = buffer[ai];
                    buffer[ai] = buffer[bi];
                    buffer[bi] += t;
                }
                offset >>= 1;
                __syncthreads();

            }
            // Write the result to the output
            ai = tid+1;
            bi = tid+1+BLOCK_SIZE/2;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            gid = global_base + tid;
            odata[gid] = buffer[ai];
            gid = global_base + tid + BLOCK_SIZE/2;
            odata[gid] = (tid < blockDim.x - 1) ? buffer[bi] : sum;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // Malloc different level of memory
            int level = (ilog2ceil(n)+7)/8;
            int** dev_ptr = new int*[level];
            int* grid_size = new int[level];
            int len = n;
            for(int i = 0; i < level; i++){
                grid_size[i] = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
                hipMalloc((void**)&dev_ptr[i], grid_size[i] * BLOCK_SIZE * sizeof(int));
                checkCUDAError("hipMalloc dev_ptr failed!");
                hipMemset(dev_ptr[i], 0, grid_size[i] * BLOCK_SIZE * sizeof(int));
                len = grid_size[i];
            }
            int* dev_tempbuff;
            int temp_size = n;
            hipMalloc((void**)&dev_tempbuff, temp_size*sizeof(int));
            checkCUDAError("hipMalloc dev_tempbuff failed!");
            hipMemcpy(dev_tempbuff, idata, temp_size*sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata failed!");

            timer().startGpuTimer();

            // Scan each block in different level
            dim3 half_block_size(BLOCK_SIZE/2);
            for(int i = 0; i < level; i++){
                kernInclusiveScanPerBlock<<<grid_size[i], half_block_size>>>(temp_size, dev_ptr[i], dev_tempbuff);
                // Gather the last element of each block
                if(i < level - 1){
                    Common::kernExtractLastElementPerBlock<<<grid_size[i+1], BLOCK_SIZE>>>( grid_size[i], 
                                                                                    BLOCK_SIZE, 
                                                                                    dev_tempbuff, 
                                                                                    dev_ptr[i]);
                }
                temp_size = grid_size[i];
            }
            // Scatter the offset to the original array
            for(int i = level - 2; i >= 0; i--){
                Common::kernAddOffset<<<grid_size[i], BLOCK_SIZE>>>(grid_size[i] * BLOCK_SIZE, dev_ptr[i], dev_ptr[i+1]);
            }

            timer().endGpuTimer();

            // Copy the result to the host
            hipMemcpy(odata, dev_ptr[0], n*sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed!");
            
            // Free the memory
            for(int i = 0; i < level; i++){
                hipFree(dev_ptr[i]);
            }
            hipFree(dev_tempbuff);
            delete[] dev_ptr;
            delete[] grid_size;
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        /* Exclusive scan in Block Level using work-efficient algorithm O(n)
        * the size of odata should be BLOCK_SIZE * gird_size, 
        * so that need to padding the last block
        * @parms
        * n: the number of blocks in idata
        * s: the stride of the block in idata
        */
        __global__ void kernInclusiveScanPerBlock(int n, int s, int* odata, const int* idata){
            int tid = threadIdx.x;
            int tid2 = tid*2;
            int global_base = blockIdx.x * BLOCK_SIZE;
            __shared__ int buffer[BLOCK_SIZE+BLOCK_SIZE/NUM_BANKS];

            // Load the data to the shared memory
            int ai = tid;
            int bi = tid + BLOCK_SIZE/2;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int gid = global_base + tid;
            int idx_in = gid * s - 1;// get last element of each pre block
            buffer[ai] = (0 < gid && gid < n) ? idata[idx_in] : 0;

            gid = global_base + tid + BLOCK_SIZE/2;
            idx_in = gid * s - 1;
            buffer[bi] = (0 < gid && gid < n) ? idata[idx_in] : 0;
            __syncthreads();
            
            // Up-Sweep (Reduce)
            int offset = 1;
            for(int stride = blockDim.x; stride > 0; stride >>= 1){
                if(tid < stride){
                    int ai = offset * (tid2 + 1) - 1;
                    int bi = offset * (tid2 + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    buffer[bi] += buffer[ai];
                }
                offset <<= 1;
                __syncthreads();
            }

            // Down-Sweep (Distribute)
            int sum = 0;
            if(tid == blockDim.x - 1){
                ai = BLOCK_SIZE - 1 + CONFLICT_FREE_OFFSET(BLOCK_SIZE - 1);
                sum = buffer[ai];
                buffer[ai] = 0;
            }
            __syncthreads();

            offset = blockDim.x;
            for(int stride = 1; stride <= blockDim.x; stride <<= 1){
                if(tid < stride){
                    int ai = offset * (tid2 + 1) - 1;
                    int bi = offset * (tid2 + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    int t = buffer[ai];
                    buffer[ai] = buffer[bi];
                    buffer[bi] += t;
                }
                offset >>= 1;
                __syncthreads();

            }
            // Write the result to the output
            ai = tid+1;
            bi = tid+1+BLOCK_SIZE/2;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            gid = global_base + tid;
            odata[gid] = buffer[ai];
            gid = global_base + tid + BLOCK_SIZE/2;
            odata[gid] = (tid < blockDim.x - 1) ? buffer[bi] : sum;
        }

        void scanOnDevice(int n, int*dev_odata,const int*dev_idata){

            // Malloc different level of memory
            int level = (ilog2ceil(n) + 7) / 8;
            int** dev_ptr = new int* [level];
            int* grid_size = new int[level];
            grid_size[0] = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
            dev_ptr[0] = dev_odata;
            for(int i = 1; i < level; i++){
                grid_size[i] = (grid_size[i-1] + BLOCK_SIZE - 1) / BLOCK_SIZE;
                hipMalloc((void**)&dev_ptr[i], grid_size[i] * BLOCK_SIZE * sizeof(int));
			    checkCUDAError("hipMalloc dev_ptr failed!");
                hipMemset(dev_ptr[i], 0, grid_size[i] * BLOCK_SIZE * sizeof(int));
            }

            // Scan each block in different level
            dim3 half_block_size(BLOCK_SIZE/2);
            kernInclusiveScanPerBlock << <grid_size[0], half_block_size >> > (n, 1, dev_odata, dev_idata);
            for(int i = 1; i < level; i++){
                kernInclusiveScanPerBlock<<<grid_size[i], half_block_size>>>(grid_size[i-1], BLOCK_SIZE, dev_ptr[i], dev_ptr[i-1]);
            }

            // Scatter the offset to the original array
            for(int i = level - 1; i > 0; i--){
                Common::kernAddOffset<<<grid_size[i-1], BLOCK_SIZE>>>(grid_size[i-1] * BLOCK_SIZE, dev_ptr[i-1], dev_ptr[i]);
            }

            // Free the memory
            for(int i = 1; i < level; i++){
                hipFree(dev_ptr[i]);
            }
            delete[] dev_ptr;
            delete[] grid_size;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata, *dev_odata;
            int size = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
            size *= BLOCK_SIZE;
            hipMalloc((void**)&dev_idata, size * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemset(dev_idata, 0, size * sizeof(int));
            hipMalloc((void**)&dev_odata, size * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMemset(dev_odata, 0, size * sizeof(int));

            hipMemcpy(dev_idata, idata, n*sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata failed!");

            timer().startGpuTimer();
            scanOnDevice(size, dev_odata, dev_idata);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, n*sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed!");

            hipFree(dev_idata);
            hipFree(dev_odata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int count = 0;
            int final_bool = 0;
            int* dev_idata, *dev_bools, *dev_indices;
            int* dev_odata;

            int gird_size = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
            int size = gird_size * BLOCK_SIZE;
            // Malloc the memory
            hipMalloc((void**)&dev_idata, size * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemset(dev_idata, 0, size * sizeof(int));

            hipMalloc((void**)&dev_bools, size * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMemset(dev_bools, 0, size * sizeof(int));

            hipMalloc((void**)&dev_indices, size * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMemset(dev_indices, 0, size * sizeof(int));

            hipMalloc((void**)&dev_odata, size * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMemset(dev_odata, 0, size * sizeof(int));

            // Copy the data to the device
            hipMemcpy(dev_idata, idata, n*sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            // Map to boolean
            Common::kernMapToBoolean<<<gird_size, BLOCK_SIZE>>>(n, dev_bools, dev_idata);

            // Scan the boolean array
            scanOnDevice(size, dev_indices, dev_bools);

            // Scatter the data
            Common::kernScatter<<<gird_size, BLOCK_SIZE>>>(n, dev_odata, dev_idata, dev_bools, dev_indices);
            timer().endGpuTimer();

            // Copy the data back to the host
            hipMemcpy(odata, dev_odata, n*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&count, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&final_bool, dev_bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            count += final_bool;
            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_indices);
            hipFree(dev_odata);
            return count;
        }
    }
}
